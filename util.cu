
#include <hip/hip_runtime.h>
#include <iostream>

void checkError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      std::cerr << msg << ": " << hipGetErrorString(err)
                << " in file " << __FILE__
                << " at line " << __LINE__ << std::endl;
      exit(EXIT_FAILURE);
  }
}