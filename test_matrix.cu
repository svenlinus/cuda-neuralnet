#include "string.h"
#include "util.h"
#include "matrix.h"
#include <stdio.h>
#include <string.h>

void test_matrixMult() {
  // (4,2)*(2,3) = (4,3)
  float a[8] = {-2,-1,
                 0, 1,
                 2, 3,
                 4, 5};
  float b[6] = {-1,0,1,
                 0,1,0};
  Matrix *A, *B, *C;
  initMatrix(&A, 4, 2);
  initMatrix(&B, 2, 3);
  initMatrix(&C, 4, 3);
  setDeviceMatrixData(A, a, 8);
  setDeviceMatrixData(B, b, 6);

  deviceMatrixMult(A, B, C, 12);

  float c[12];
  getDeviceMatrixData(c, C, 12);

  char result[64];
  char expected[64] = "2 -1 -2 0 1 0 -2 3 2 -4 5 4";
  int offset = 0;
  for (int i = 0; i < 12; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix mult\n");
  printf("Result: %s\n", result);
  printf("Expect: %s\n", expected);
  if (strncmp(result, expected, 27) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");

  freeMatrix(&A);
  freeMatrix(&B);
  freeMatrix(&C);
}

void test_matrixElementWise() {
  Matrix *A, *B, *C;
  float data[6] = {0,1,2,3,4,5};

  initMatrix(&A, 2, 3);
  initMatrix(&B, 2, 3);
  initMatrix(&C, 2, 3);
  setDeviceMatrixData(A, data, 6);
  setDeviceMatrixData(B, data, 6);

  deviceMatrixAdd(A, B, C, 1, 6);

  float c[6];
  getDeviceMatrixData(c, C, 6);

  char result[64];
  int offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix add\n");
  printf("Result: %s\n", result);
  printf("Expect: 0 2 4 6 8 10\n");
  if (strncmp(result, "0 2 4 6 8 10", 12) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }

  deviceMatrixAdd(A, B, C, -1, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix sub\n");
  printf("Result: %s\n", result);
  printf("Expect: 0 0 0 0 0 0\n");
  if (strncmp(result, "0 0 0 0 0 0", 11) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }

  deviceHadamardProd(A, B, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing hadamardProd \n");
  printf("Result: %s\n", result);
  printf("Expect: 0 1 4 9 16 25\n");
  if (strncmp(result, "0 1 4 9 16 25", 13) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }

  deviceSigmoid(C, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%.2f ", c[i]);
  }
  printf("Testing sigmoid \n");
  printf("Result: %s\n", result);
  printf("Expect: 0.50 0.73 0.98 1.00 1.00 1.00\n");
  if (strncmp(result, "0.50 0.73 0.98 1.00 1.00 1.00", 28) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");


  deviceMatrixScale(A, 2, C, 6);
  getDeviceMatrixData(c, C, 6);

  offset = 0;
  for (int i = 0; i < 6; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%.2f ", c[i]);
  }
  printf("Testing scalar \n");
  printf("Result: %s\n", result);
  printf("Expect: 0 2 4 6 8 10\n");
  if (strncmp(result, "0 2 4 6 8 10", 28) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");

  freeMatrix(&A);
  freeMatrix(&B);
  freeMatrix(&C);
}

void test_transpose() {
  Matrix *A, *tA, *result;
  float a[8] = {
    0,1,2,3,
    4,5,6,7
  };
  initMatrix(&A, 2, 4);     // A (2,4)
  setDeviceMatrixData(A, a, 8);
  matrixTranpose(A, &tA);   // tA (4,2)
  hipDeviceSynchronize();
  checkError("Transpose");

  initMatrix(&result, 2, 2);
  deviceMatrixMult(A, tA, result, 4);  // (2,4)(4,2) = (2,2)

  float c[12];
  getDeviceMatrixData(c, result, 4);

  char result[32];
  char expected[32] = "14 38 38 126";
  int offset = 0;
  for (int i = 0; i < 12; ++i) {
    offset += snprintf(result + offset, sizeof(result) - offset, "%d ", (int)c[i]);
  }
  printf("Testing matrix mult\n");
  printf("Result: %s\n", result);
  printf("Expect: %s\n", expected);
  if (strncmp(result, expected, 12) != 0) {
    printf("FAILED\n");
    exit(EXIT_FAILURE);
  }
  printf("\nPASSED\n\n");

  freeMatrix(&A);
  freeMatrix(&tA);
  freeMatrix(&result);
}

int main() {

  test_matrixMult();
  test_matrixElementWise();
  test_transpose();

  return 0;
}